
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){

  int a, *d_a;
  hipMalloc(&d_a, sizeof(d_a[0]));
  hipMemcpy(d_a, &a, sizeof(a), hipMemcpyHostToDevice);
  hipStream_t str;
  hipStreamCreate(&str);
  hipMemcpyAsync(d_a, &a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_a, &a, sizeof(a), hipMemcpyHostToDevice, str);
  hipDeviceSynchronize();
}