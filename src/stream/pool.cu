#include "pool.h"

StreamPool::StreamPool(int initial_size) {
    for(int i = 0; i < initial_size; i++) {
        hipStream_t stream;
        hipStreamCreate(&stream);
        available_streams.push(stream);
    }
}

StreamPool::~StreamPool() {
    while(!available_streams.empty()) {
        hipStream_t stream = available_streams.front();
        available_streams.pop();
        hipStreamDestroy(stream);
    }
}

hipStream_t StreamPool::getStream() {
    std::lock_guard<std::mutex> lock(mutex);
    if (available_streams.empty()) {
        hipStream_t stream;
        hipStreamCreate(&stream);
        available_streams.push(stream);
    }
    hipStream_t stream = available_streams.front();
    available_streams.pop();
    return stream;
}

void StreamPool::returnStream(hipStream_t stream) {
    std::lock_guard<std::mutex> lock(mutex);
    available_streams.push(stream);
}
